#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <time.h>

#define K 5
#define NUM_THREADS 32
#define NUM_BLOCKS	8
#define MAX_ITER	1

hipError_t kmeansWithCuda();

__device__ int distance(int x1, int x2) {
	return sqrtf((x2 - x1)*(x2 - x1));
}

__global__ void kmeansClusterAssignmentKernel(int *d_dataPoints, int *d_clusterAssignment, int *d_centroids, int N){
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= N) return;

	int min_dist = 5000;
	int closest_centroid = 0;

	for (int c = 0; c < K; c++) {
		int dist = distance(d_dataPoints[idx], d_centroids[c]);
		if (dist < min_dist) {
			min_dist = dist;
			closest_centroid = c;
		}
	}

	d_clusterAssignment[idx] = closest_centroid;
	//printf("idx du kernel : %d \n", idx);
}

/* Non fonctionnelle pour le moment */
__global__ void kmeansCentroidUpdate(int *d_dataPoints, int *d_clusterAssignment, int *d_centroids, int *d_clustersSize, int N) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= N) return;	

	const int s_idx = threadIdx.x;
	__shared__ int s_dataPoints[32];
	s_dataPoints[s_idx] = d_dataPoints[idx];
	__shared__ int s_clusterAssignment[32];
	s_clusterAssignment[s_idx] = d_clusterAssignment[s_idx];

	__syncthreads();

	//On somme par l'indice 0 pour chaque bloc
	if (s_idx == 0) {
		int block_clusterDataPointsSum[K] = { 0 };
		int block_clustersSizes[K] = { 0 };

		//sommation
		for (int i = 0; i < blockDim.x; i++) {
			int clusterId = s_clusterAssignment[i];
			block_clusterDataPointsSum[clusterId] += s_dataPoints[i];
			block_clustersSizes[clusterId] += 1;
		}

		for (int j = 0; j < K; j++) {
			printf("atomicAdd, valeur de la centroide avant addition : %d \n", &d_centroids[j]);
			printf("valeur de la somme avant addition : %d \n", block_clusterDataPointsSum[j]);
			atomicAdd(&d_centroids[j], block_clusterDataPointsSum[j]);
			printf("valeur apres addition de centroide : %d \n", &d_centroids[j]);
			atomicAdd(&d_clustersSize[j], block_clustersSizes[j]);
		}
	}

	__syncthreads();

	if (idx < K) {
		d_centroids[idx] = d_centroids[idx] / d_clustersSize[idx];
	}

}

int main(){
    // Add vectors in parallel.
    hipError_t cudaStatus = kmeansWithCuda();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kmeansWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

std::string readFile(const std::string &toRead) {
	std::ifstream fd(toRead.c_str());
	std::string buffer;
	char ch;
	while (fd.get(ch)) buffer.push_back(ch);
	buffer.push_back('\n');
	return buffer;
}

std::vector<int> parseFile(const std::string &data) {
	std::vector<int> res;
	std::string tmp;

	for (int i = 0; i < data.length(); i++) {
		//std::cout << "i : " << i << " et fileData[i] : " << data[i] << std::endl;
		if (data[i] == ' ' || data[i] == '\n') {
			if (tmp.empty())	continue;
			res.push_back(std::stoi(tmp));
			tmp.erase();
		}
		else {
			tmp.push_back(data[i]);
		}
	}
	return res;
}

int* vector2int(const std::vector<int>& v, const int& size) {
	int *res = new int[size];
	for (int i = 0; i < size; i++) {
		res[i] = v[i];
	}
	return res;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t kmeansWithCuda(){
	std::ifstream file;
	std::string path = "C:/Users/Quentin/Desktop/Le Reste/ProjetENSTAHusky/carteDistance.txt";
	std::string fileData;
	srand(time(NULL));
    int *dev_dataPoints = 0;
    int *dev_clustersAssigned = 0;
    int *dev_centroids = 0;
	int *dev_clustersSize = 0;
	float millisecondsKernel = 0, millisecondsGlobal = 0;
	int centroids[K], clustersSize[K];
	for (int i = 0; i < K; i++) {
		centroids[i] = rand() % 200 + i*200;
		std::cout << "Centroide " << i << " : " << centroids[i] << std::endl;
		clustersSize[i] = 0;
	}
	int currentIter = 1;
    hipError_t cudaStatus;
	hipEvent_t startKernel, stopKernel, startGlobal, stopGlobal;
	hipEventCreate(&startKernel);
	hipEventCreate(&startGlobal);
	hipEventCreate(&stopKernel);
	hipEventCreate(&stopGlobal);

	// Récupération des données provenant d'un fichier txt d'entrée
	fileData = readFile(path);
	//std::cout << fileData << std::endl;
	std::cout << "nombre d'elements: " << fileData.length() << std::endl;
	std::vector<int> res = parseFile(fileData);
	int *resInt = vector2int(res, res.size());
	int *clusterAssigned = (int*)malloc(res.size() * sizeof(int));
	//for(size_t i = 0; i < res.size(); i++)	std::cout << res[i] << std::endl;
	//std::cout << "nombre d'elements dans le vecteur : " << res.size() << std::endl;
	std::cout << "Nombre d'elements du vecteur apres parsing : " << res.size() << std::endl;
	std::cout << "premier element du vecteur : " << res[0] << std::endl;
	std::cout << "premier element du tableau de int : " << resInt[0] << std::endl;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_clustersSize, K * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_clustersSize - hipMalloc failed! \n");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_centroids, K * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_centroids - hipMalloc failed! \n");
		goto Error;
	}

    cudaStatus = hipMalloc((void**)&dev_dataPoints, res.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "resInt - hipMalloc failed! \n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_clustersAssigned, res.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_clustersAssigned - hipMalloc failed! \n");
        goto Error;
    }
	
	std::cout << "Mallocs sur le device termines" << std::endl;
	hipEventRecord(startGlobal);
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_dataPoints, resInt, res.size() * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hostToDevice - dataPoints - hipMemcpy failed! \n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_centroids, &centroids, K * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hostToDevice - centroids - hipMemcpy failed! \n");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_clustersSize, clustersSize, K * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hostToDevice - clusterSize - hipMemcpy failed! \n");
		goto Error;
	}

	std::cout << "Memcpy vers le device termines" << std::endl;


	hipEventRecord(startKernel);
	while (currentIter < MAX_ITER) {
		// Launch a kernel on the GPU with one thread for each element.
		std::cout << "Lancement du kernel Cluster Assignment" << std::endl;
		kmeansClusterAssignmentKernel <<< (res.size() + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (dev_dataPoints, dev_clustersAssigned, dev_centroids, res.size());
		std::cout << "Kernel Cluster Assignment applique" << std::endl;

/*		cudaStatus = hipMemcpy(centroids, dev_centroids, K * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "clusterAssigned - hipMemcpy failed! \n");
			goto Error;
		}
		for (int i = 0; i < K; i++) {
			std::cout << "Iteration " << currentIter <<", centroide " << i << " situe a une distance " << centroids[i] << std::endl;
		}

		hipMemset(dev_centroids, 0, K * sizeof(int));
		hipMemset(dev_clustersSize, 0, K * sizeof(int));
		std::cout << "Lancement du kernel Centroid Update" << std::endl;
		kmeansCentroidUpdate <<< (res.size() + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS >> > (dev_dataPoints, dev_clustersAssigned, dev_centroids, dev_clustersSize, res.size());
		std::cout << "Kernel Centroid Update applique" << std::endl;*/
		currentIter++;
	}
	hipEventRecord(stopKernel);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kmeansClusterAssignmentKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kmeansClusterAssignmentKernel!\n", cudaStatus);
        goto Error;
    }

	std::cout << "Avant memcpy de dev_cluster vers l'host" << std::endl;
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(clusterAssigned, dev_clustersAssigned, res.size() * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "clusterAssigned - hipMemcpy failed! \n");
        goto Error;
    }
	std::cout << "Memcpy de dev_cluster vers l'host termine" << std::endl;

	hipEventRecord(stopGlobal);
	std::cout << "Memcpy vers l'host termines" << std::endl;
/*	for (int i = 0; i < res.size(); i++) {
		std::cout << "Clusters assignes pour i valant : " << i << " cluster : " << clusterAssigned[i] << std::endl;
	}*/

	hipEventSynchronize(stopKernel);
	hipEventSynchronize(stopGlobal);
	hipEventElapsedTime(&millisecondsKernel, startKernel, stopKernel);
	hipEventElapsedTime(&millisecondsGlobal, startGlobal, stopGlobal);
	
	printf("Performances du kernel : %f msec \n\r", millisecondsKernel);
	printf("Performances du kernel, avec transferts memoire : %f msec \n\r", millisecondsGlobal);

Error:
    hipFree(dev_clustersAssigned);
    hipFree(dev_dataPoints);
    hipFree(dev_centroids);
	free(clusterAssigned);

    return cudaStatus;
}
